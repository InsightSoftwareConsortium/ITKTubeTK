/*=========================================================================
 *
 *  Copyright Insight Software Consortium
 *
 *  Licensed under the Apache License, Version 2.0 ( the "License" );
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
*=========================================================================*/
#ifndef __itktubeGPUArrayFireGaussianDerivativeFilter_hxx
#define __itktubeGPUArrayFireGaussianDerivativeFilter_hxx

#include "itktubeGPUArrayFireGaussianDerivativeFilter.h"
#include "itktubeGaussianDerivativeImageSource.h"
#include "itktubePadImageFilter.h"
#include "itktubeRegionFromReferenceImageFilter.h"

#include "itkParametricImageSource.h"
#include "itkSize.h"

namespace itk
{

namespace tube
{
//----------------------------------------------------------------------------
template< typename TInputImage, typename TOutputImage >
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::GPUArrayFireGaussianDerivativeFilter()
{
  m_PaddedInputImage = NULL;
  m_PaddedKernelImage = NULL;
  m_ConvolvedImage = NULL;
  m_PaddedTotalSize = 0;
  //m_Plan;

  m_GaussSource = GaussianDerivativeImageSourceType::New();

  m_ComplexPaddedInputImage = NULL;
  m_OnGPUPaddedInputImage = NULL;
  m_ComplexPaddedKernelImage = NULL;
  m_OnGPUPaddedKernelImage = NULL;

  m_LastInputImage = NULL;
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::ComputeInputImageFFT()
{
  typedef PadImageFilter< InputImageType, RealImageType >
  PadFilterType;
  typename PadFilterType::Pointer padFilter = PadFilterType::New();
  padFilter->SetInput( this->GetInput() );
  padFilter->SetGreatestPrimeFactor( 5 );
  padFilter->SetPadMethod( PadFilterType::ZERO_FLUX_NEUMANN );
  padFilter->Update();

  m_PaddedInputImage = padFilter->GetOutput();

  typename InputImageType::SizeType paddedSize = 
    m_PaddedInputImage.GetSize();
  m_PaddedTotalSize = 1;
  for( unsigned int d=0; d<ImageDimension; ++d )
    {
    m_PaddedTotalSize *= paddedSize[d];
    }

  //
  // Allocate complex image
  //
  if( m_ComplexPaddedInputImage != NULL )
    {
    delete m_ComplexPaddedInputImage;
    }
  m_ComplexPaddedInputImage = reinterpret_cast<Complex *>(
    malloc(sizeof(Complex) * m_PaddedTotalSize) );
  itk::ImageRegionIterator<InputImageType> iter( m_PaddedInputImage,
    m_PaddedInputImage->GetLargestPossibleRegion() );
  unsigned int count = 0;
  while( !iter.IsAtEnd() )
    {
    m_ComplexPaddedInputImage[count].x = iter.Get();
    m_ComplexPaddedInputImage[count].y = 0;
    ++iter;
    ++count;
    }

  //
  // Allocate complex kernel
  //
  if( m_ComplexPaddedKernelImage != NULL )
    {
    delete m_ComplexPaddedKernelImage;
    }
  m_ComplexPaddedKernelImage = reinterpret_cast<Complex *>(
    malloc(sizeof(Complex) * m_PaddedTotalSize) );

  // 
  // Identify NVidia device
  //
  int deviceCount = 0;
  checkCudaErrors( hipGetDeviceCount(&deviceCount) );

  int targetDevice = 0;
  targetDevice = findCudaDevice();
  cudaCheckErrors( hipSetDevice(targetDevice) );

  //
  // Allocate image memory on GPU
  //
  int mem_size = sizeof(Complex) * m_PaddedTotalSize;

  if( m_OnGPUPaddedInputImage != NULL )
    {
    delete m_OnGPUPaddedInputImage;
    }
  checkCudaErrors( hipMalloc(
    reinterpret_cast<void **>(&m_OnGPUPaddedInputImage), mem_size));

  //
  // Copy image to GPU
  //
  checkCudaErrors( hipMemcpy(
    m_OnGPUPaddedInputImage, m_ComplexPaddedInputImage, mem_size,
    hipMemcpyHostToDevice));

  //
  // Describe GPU workspace for image
  //
  switch( ImageDimension )
    {
    case 1:
      checkCudaErrors(hipfftPlan1d(&m_Plan, inputSize[0],
          HIPFFT_C2C, 1));
      break;
    case 2:
      checkCudaErrors(hipfftPlan2d(&m_Plan, inputSize[1],
          inputSize[0], HIPFFT_C2C));
      break;
    case 3:
      checkCudaErrors(hipfftPlan3d(&m_Plan, inputSize[2],
          inputSize[1], inputSize[0], HIPFFT_C2C));
      break;
    default:
      itk::InvalidArgumentError e( __FILE__, __LINE__ );
      e.SetDescription( "Only Dimensions up to 3 are supported" );
      e.SetLocation( "CudaFFT" );
      throw e;
    }
  
  //
  // Perform FFT of image
  //
  checkCudaErrors(hipfftExecC2C(m_Plan,
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedInputImage),
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedInputImage),
    HIPFFT_FORWARD));

  //
  // Setup Gaussian kernel
  //
  const typename RealImageType::RegionType inputRegion(
    this->GetInput()->GetLargestPossibleRegion() );
  const typename RealImageType::SizeType inputSize
    = inputRegion.GetSize();

  const typename RealImageType::RegionType fftRegion(
    m_PaddedInputImage->GetLargestPossibleRegion() );
  const typename RealImageType::SizeType fftSize
    = fftRegion.GetSize();
  const typename RealImageType::SpacingType fftSpacing =
    m_PaddedInputImage->GetSpacing();
  const typename RealImageType::PointType fftOrigin =
    m_PaddedInputImage->GetOrigin();
  const typename RealImageType::DirectionType fftDirection =
    m_PaddedInputImage->GetDirection();

  m_GaussSource->SetIndex( fftRegion.GetIndex() );
  m_GaussSource->SetSize( fftSize );
  m_GaussSource->SetSpacing( fftSpacing );
  m_GaussSource->SetOrigin( fftOrigin );
  m_GaussSource->SetDirection( fftDirection );

  typename GaussianDerivativeImageSourceType::PointType mean;
  typename GaussianDerivativeImageSourceType::IndexType meanIndex;
  for( unsigned int ii = 0; ii < ImageDimension; ++ii )
    {
    const int halfLength = ( inputSize[ii]  / 2.0 );
    meanIndex[ii] = inputRegion.GetIndex() [ii] + halfLength;
    }
  this->GetInput()->TransformIndexToPhysicalPoint( meanIndex,
    mean );
  m_GaussSource->SetMean( mean );
}


template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::ComputeKernelImageFFT()
{
  // 
  // Define kernel
  //
  m_GaussSource->SetSigmas( this->m_Sigmas );
  m_GaussSource->SetOrders( this->m_Orders );
  m_GaussSource->Update();

  typename FFTShiftFilterType::Pointer fftShiftFilter =
    FFTShiftFilterType::New();
  fftShiftFilter->SetInput( m_GaussSource->GetOutput() );
  fftShiftFilter->Update();

  m_PaddedKernelImage = fftShiftFilter->GetOutput();

  itk::ImageRegionIterator<InputImageType> iter(
    m_PaddedKernelImage,
    m_PaddedKernelImage->GetLargestPossibleRegion() );
  unsigned int count = 0;
  while( !iter.IsAtEnd() )
    {
    m_ComplexPaddedKernelImage[count].x = iter.Get();
    m_ComplexPaddedKernelImage[count].y = 0;
    ++iter;
    ++count;
    }

  //
  // Identify NVidia device
  //
  int deviceCount = 0;
  checkCudaErrors( hipGetDeviceCount(&deviceCount) );

  int targetDevice = 0;
  targetDevice = findCudaDevice();
  cudaCheckErrors( hipSetDevice(targetDevice) );

  //
  // Allocate GPU memory for kernel and copy kernel to GPU
  //
  int mem_size = sizeof(Complex) * m_PaddedTotalSize;
  if( m_OnGPUPaddedKernelImage == NULL )
    {
    checkCudaErrors( hipMalloc(
      reinterpret_cast<void **>(&m_OnGPUPaddedKernelImage),
      mem_size));
    }
  checkCudaErrors( hipMemcpy(
    m_OnGPUPaddedKernelImage, m_ComplexPaddedKernelImage, mem_size,
    hipMemcpyHostToDevice));

  //
  // Describe GPU workspace for kernel
  //
  hipfftHandle plan_adv;
  size_t workSize;
  long long int new_size_long[3];
  for( unsigned int d=0; d<ImageDimension; ++d)
    {
    new_size_long[d] = paddedSize[d];
    }
  checkCudaErrors(hipfftCreate(&plan_adv));
  switch( ImageDimension )
    {
    case 1:
      checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 1,
          new_size_long, NULL, 1, 1, HIP_C_32F,
          NULL, 1, 1, HIP_C_32F, 1, &workSize, HIP_C_32F));
      break;
    case 2:
      checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 2,
          &new_size_long, NULL, 1, 1, HIP_C_32F,
          NULL, 1, 1, HIP_C_32F, 1, &workSize, HIP_C_32F));
      break;
    case 3:
      checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 3,
          &new_size_long, NULL, 1, 1, HIP_C_32F,
          NULL, 1, 1, HIP_C_32F, 1, &workSize, HIP_C_32F));
      break;
    default:
      itk::InvalidArgumentError e( __FILE__, __LINE__ );
      e.SetDescription( "Only Dimensions up to 3 are supported" );
      e.SetLocation( "CudaFFT" );
      throw e;
    }

  //
  // Perform FFT of kernel
  //
  checkCudaErrors(hipfftExecC2C(plan_adv,
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedKernelImage),
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedKernelImage),
    HIPFFT_FORWARD));
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::ComputeConvolvedImageFFT()
{
  //
  // Allocate GPU memory for convolution results and init memory
  //
  if( m_OnGPUPaddedConvolutionImage == NULL )
    {
    checkCudaErrors( hipMalloc(
      reinterpret_cast<void **>(&m_OnGPUPaddedConvolutionImage),
      mem_size));
    }

  //
  // Compute convolution
  //
  GPUFFTComplexPointwiseMulAndScale<<<32, 256>>>(
    m_OnGPUPaddedConvolutionImage, m_OnGPUPaddedInputImage,
    m_OnGPUPaddedKernelImage, m_PaddedTotalSize,
    1.0f / m_PaddedTotalSize);
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::ComputeConvolvedImage()
{
  checkCudaErrors(hipfftExecC2C( m_Plan,
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedConvolutionImage),
    reinterpret_cast<hipfftComplex *>(m_OnGPUPaddedConvolutionImage),
    HIPFFT_BACKWARD));

  Complex * tmpComplexConvolutionImage =
    reinterpret_cast<Complex *>(
      malloc(sizeof(Complex) * m_PaddedTotalSize) );
  checkCudaErrors( hipMemcpy(
    tmpComplexConvolutionImage, m_OnGPUPaddedConvolutionImage,
    mem_size, hipMemcpyDeviceToHost));

  m_ConvolvedImage = TOutputImage::New();
  m_ConvolvedImage->SetRegions(
    m_InputImage->GetLargestPossibleRegion() );
  m_ConvolvedImage->CopyInformation( m_InputImage );
  m_ConvolvedImage->Allocate();
  itk::ImageRegionIterator<OutputImageType> iterPad(
    m_PaddedInputImage,
    m_PaddedInputImage->GetLargestPossibleRegion() );
  itk::ImageRegionIterator<OutputImageType> iter(
    m_ConvolvedImage,
    m_ConvolvedImage->GetLargestPossibleRegion() );
  unsigned int count = 0;
  while( !iter.IsAtEnd() )
    {
    if( iterPad.GetIndex() == iter.GetIndex() )
      {
      iter.Set( tmpComplexConvolutionImage[count].x );
      ++iter;
      }
    ++iterPad;
    ++count;
    }

  delete tmpComplexConvolutionImage;
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::GenerateData()
{
  if( m_LastInputImage != this->GetInput() )
    {
    m_LastInputImage = this->GetInput();
    ComputeInputImageFFT();
    }

  ComputeKernelImageFFT();
  ComputeConvolvedImageFFT();
  ComputeConvolvedImage();

  this->SetNthOutput( 0, m_ConvolvedImage );
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::GenerateNJet( typename TOutputImage::Pointer & D,
  std::vector< typename TOutputImage::Pointer > & dX,
  std::vector< typename TOutputImage::Pointer > & dXX )
{
  if( m_LastInputImage != this->GetInput() )
    {
    m_LastInputImage = this->GetInput();
    ComputeInputImageFFT();
    }

  if( dX.size() != ImageDimension )
    {
    dX.resize( ImageDimension );
    }

  unsigned int ddxSize = 0;
  for( unsigned int i = 1; i<=ImageDimension; ++i )
    {
    ddxSize += i;
    }
  if( dXX.size() != ddxSize )
    {
    dXX.resize( ddxSize );
    }

  // Compute G_0
  this->m_Orders.Fill( 0 );

  this->ComputeKernelImageFFT();
  this->ComputeConvolvedImageFFT();
  this->ComputeConvolvedImage();
  D = m_ConvolvedImage;

  // Compute G_1
  std::vector<Complex *> dXKernelImageFFT( ImageDimension );
  for( unsigned int i = 0; i<ImageDimension; ++i )
    {
    this->m_Orders[i] = 1;
    this->ComputeKernelImageFFT();
    this->ComputeConvolvedImageFFT();
    this->ComputeConvolvedImage();
    dXKernelImageFFT[i] = m_OnGPUPaddedKernelImage;
    m_OnGPUPaddedKernelImage = NULL;
    dX[i] = m_ConvolvedImage;
    this->m_Orders[i] = 0;
    }

  // compute G_2
  unsigned int count = 0;
  for( unsigned int i = 0; i<ImageDimension; ++i )
    {
    m_OnGPUPaddedKernelImage = dXKernelImageFFT[i];
    this->ComputeConvolvedImageFFT();
    Complex * tmpConvolvedFFT = m_OnGPUPaddedConvolvedImage;
    Complex * tmpInputFFT = m_OnGPUPaddedInputImage;
    m_OnGPUPaddedConvolvedImage = NULL;
    this->m_Orders[i] = 1;
    for( unsigned int j = i; j<ImageDimension; ++j )
      {
      this->m_Orders[j] = 1;
      m_OnGPUPaddedInputImage = tmpConvolvedFFT;
      m_OnGPUPaddedKernelImage = dXKernelImageFFT[j];
      this->ComputeConvolvedImageFFT();
      this->ComputeConvolvedImage();
      dXX[ count++ ] = m_ConvolvedImage;
      this->m_Orders[j] = 0;
      }
    hipFree( tmpConvolvedFFT );
    m_OnGPUPaddedInputImage = tmpInputFFT;
    this->m_Orders[i] = 0;
    }

  delete * dXKernelImgeFFT;

  this->SetNthOutput( 0, D );
}

template< typename TInputImage, typename TOutputImage >
void
GPUArrayFireGaussianDerivativeFilter<TInputImage, TOutputImage>
::PrintSelf( std::ostream & os, Indent indent ) const
{
  this->Superclass::PrintSelf( os, indent );

  if( m_ConvolvedImage.IsNotNull() )
    {
    os << indent << "Convolved Image   : " << m_ConvolvedImage << std::endl;
    }
  else
    {
    os << indent << "Convolved Image   : NULL" << std::endl;
    }

  if( m_LastInputImage != NULL )
    {
    os << indent << "Last Input Image    : Initialized" << std::endl;
    }
  else
    {
    os << indent << "Last Input Image    : NULL" << std::endl;
    }
}

} // End namespace tube

} // End namespace itk

#endif
